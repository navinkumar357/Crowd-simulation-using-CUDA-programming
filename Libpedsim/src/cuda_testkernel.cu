#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuda_testkernel.h"
#include <stdio.h>
#include <math.h>
#include "ped_model.h"
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t computeNextPositionWithCuda(const int *x, const int *y, const float *destinationX, const float *destinationY, unsigned int size, int *desiredX, int *desiredY);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void computeNextPositionKernel(int *desiredX, int *desiredY, int *x, int *y, float *destinationX, float *destinationY)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	float diffX = destinationX[i] - x[i];
	float diffY = destinationY[i] - y[i];
	float len = sqrt(diffX * diffX + diffY * diffY);

	float desiredXFloat = round(x[i] + diffX / len);
	float desiredYFloat = round(y[i] + diffY / len);

	desiredX[i] = int(desiredXFloat);
	desiredY[i] = int(desiredYFloat);
}

__global__ void fadeHeatmapKernel(int *heatmap, int size)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int row = 0; row < size; row++) {
		heatmap[row * size + col] = (int)round(heatmap[row * size + col] * 0.8);
	}
}

__global__ void updateHeatmapKernel(int *heatmap, int size, int *desiredX, int *desiredY, int agents)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx > agents) return;
	if (desiredX[idx] < 0 || desiredX[idx] >= size || desiredY[idx] < 0 || desiredY[idx] >= size)
	{
		return;
	}

	atomicAdd(heatmap + desiredY[idx] * size + desiredX[idx], 40);
	atomicMin(heatmap + desiredY[idx] * size + desiredX[idx], 255);
}

/*__global__ void scaleHeatmapKernel(int *heatmap, int *scaled_heatmap, int size, int cell_size)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int start = threadId % size;
	int step = threadId / size;
	start += step * size * cell_size;
	int value = heatmap[threadId];
	for (int cellY = 0; cellY < cell_size; cellY++)
	{
		for (int cellX = 0; cellX < cell_size; cellX++)
		{
			int index = start * cell_size + cellX;
			index += size * cell_size * cellY;
			scaled_heatmap[index] = value;
		}
	}
}*/

__global__ void blurHeatmapKernel(int *heatmap, int *blurred_heatmap, int size, int cell_size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int row = blockIdx.x;
	int j = 0;
	// Block size 
	// num of rows per block = 5 + 4 = 9
	int thread_id = threadIdx.x;
	__shared__ int hm_s[SIZE * 2];
	// store two rows of the heatmap inide shared memory

	for (int i = 0; i < 2; i += 1) {
		hm_s[thread_id + i * SIZE] = heatmap[thread_id + (row + i) * SIZE] < 255 ? heatmap[thread_id + (row + i) * SIZE] : 255;
		heatmap[thread_id + (row + i) * SIZE] = hm_s[thread_id + i * SIZE];
	}
	// synchronize threads
	__syncthreads();
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
	{ 4, 16, 26, 16, 4 },
	{ 7, 26, 41, 26, 7 },
	{ 4, 16, 26, 16, 4 },
	{ 1, 4, 7, 4, 1 }
	};
	#define WEIGHTSUM 273
	// every thread calculates 25 values of the scaled heatmap
	// first row 
	if ((thread_id != 0) & (thread_id != 1023)) {
		for (int i = 0; i < 5; i++) {
			int sum = 0;
			int index;
			for (int k = -2; k < 3; k++) {
				for (int l = -2; l < 3; l++) {
					index = (int)((thread_id * 5 + i + k) / 5);
					sum += hm_s[index] * w[2 + k][2 + l];
				}
			}
			int value = sum / WEIGHTSUM;
			blurred_heatmap[thread_id * 5 + i + (row * 5 * SCALED_SIZE)] = 0x00FF0000 | value << 24;
		}
		for (int j = 2; j < 6; j++) {
			for (int i = 0; i < 5; i++) {
				int index;
				int sum = 0;
				for (int k = -2; k < 3; k++) {
					for (int l = -2; l < 3; l++) {
						index = (int)((thread_id * 5 + i + k) / 5) + SIZE * (int)((j + l) / 4);
						sum += hm_s[index] * w[2 + k][2 + l];
					}
				}
				int value = sum / WEIGHTSUM;
				blurred_heatmap[thread_id * 5 + i + (((row * 5) + j - 1)* SCALED_SIZE)] = 0x00FF0000 | value << 24;
			}
		}
	}

}

void cuda_setupHeatmap(int *heatmap, int *scaled_heatmap, int *blurred_heatmap)
{
	const int size = 1024;
	const int cell_size = 5;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	cudaStatus = hipHostAlloc(&heatmap, size * size * sizeof(int), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
		printf("Error allocating pinned host memory\n");
	cudaStatus = hipHostAlloc(&scaled_heatmap, size * cell_size * size * cell_size * sizeof(int), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
		printf("Error allocating pinned host memory\n");
	cudaStatus = hipHostAlloc(&blurred_heatmap, size * cell_size * size * cell_size * sizeof(int), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
		printf("Error allocating pinned host memory\n");
}

hipError_t createHeatmapWithCuda(Ped::Model *model, int *heatmap, int *scaled_heatmap, int *blurred_heatmap, const int size, const int cell_size, int *desiredX, int *desiredY, const int agents)
{
	int *dev_heatmap = 0;
	int *dev_scaled_heatmap = 0;
	int *dev_blurred_heatmap = 0;
	int *dev_desiredX = 0;
	int *dev_desiredY = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	cudaStatus = hipMalloc(&dev_heatmap, size * size * sizeof(int));
	cudaStatus = hipMalloc(&dev_scaled_heatmap, size * cell_size * size * cell_size * sizeof(int));
	cudaStatus = hipMalloc(&dev_blurred_heatmap, size * cell_size * size * cell_size * sizeof(int));
	cudaStatus = hipMalloc(&dev_desiredX, size * sizeof(int));
	cudaStatus = hipMalloc(&dev_desiredY, size * sizeof(int));

	cudaStatus = hipMemcpyAsync(dev_heatmap, heatmap, size * size * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyAsync(dev_scaled_heatmap, scaled_heatmap, size * cell_size * size * cell_size * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyAsync(dev_blurred_heatmap, blurred_heatmap, size * cell_size * size * cell_size * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyAsync(dev_desiredX, desiredX, size * sizeof(int), hipMemcpyHostToDevice); // Not pinned.
	cudaStatus = hipMemcpyAsync(dev_desiredY, desiredY, size * sizeof(int), hipMemcpyHostToDevice); // Not pinned.
	
	dim3 grid(1024, 5, 5);
	dim3 block(1024, 1, 1);


	float fade_time;
	hipEvent_t start_fade_event, stop_fade_event;
	hipEventCreate(&start_fade_event);
	hipEventCreate(&stop_fade_event);
	hipEventRecord(start_fade_event, 0);

	fadeHeatmapKernel << < 1, 1024 >> > (dev_heatmap, size);
	
	hipEventRecord(stop_fade_event, 0);
	hipEventSynchronize(stop_fade_event);
	hipEventElapsedTime(&fade_time, start_fade_event, stop_fade_event);
	hipEventDestroy(start_fade_event);
	hipEventDestroy(stop_fade_event);

	float update_hm_time;
	hipEvent_t start_update_event, stop_update_event;
	hipEventCreate(&start_update_event);
	hipEventCreate(&stop_update_event);
	hipEventRecord(start_update_event, 0);

	updateHeatmapKernel << < 1, 1024 >> > (dev_heatmap, size, dev_desiredX, dev_desiredY, agents);
	
	hipEventRecord(stop_update_event, 0);
	hipEventSynchronize(stop_update_event);
	hipEventElapsedTime(&update_hm_time, start_update_event, stop_update_event);
	hipEventDestroy(start_update_event);
	hipEventDestroy(stop_update_event);

	float blur_time;
	hipEvent_t start_blur_event, stop_blur_event;
	hipEventCreate(&start_blur_event);
	hipEventCreate(&stop_blur_event);
	hipEventRecord(start_blur_event, 0);

	blurHeatmapKernel << < 1024, 1024>> > (dev_heatmap, dev_blurred_heatmap, size, cell_size);

	hipEventRecord(stop_blur_event, 0);
	hipEventSynchronize(stop_blur_event);
	hipEventElapsedTime(&blur_time, start_blur_event, stop_blur_event);
	hipEventDestroy(start_blur_event);
	hipEventDestroy(stop_update_event);

	float collision_time;
	hipEvent_t start_collision_event, stop_collision_event;
	hipEventCreate(&start_collision_event);
	hipEventCreate(&stop_collision_event);
	hipEventRecord(start_collision_event, 0);

	model->collision_detection_regions();
	
	hipEventRecord(stop_collision_event, 0);
	hipEventSynchronize(stop_collision_event);
	hipEventElapsedTime(&collision_time, start_collision_event, stop_collision_event);
	hipEventDestroy(start_collision_event);
	hipEventDestroy(stop_collision_event);
	
	
	printf("Fade event:  %0.6f ms\n", fade_time);
	printf("Update heatmap:  %0.6f ms\n", update_hm_time);
	printf("Blur heatmap:  %0.6f ms\n", blur_time);
	printf("Collision Detection:  %0.6f ms\n", collision_time);

	cudaStatus = hipMemcpyAsync(heatmap, dev_heatmap, size * size * sizeof(int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpyAsync(scaled_heatmap, dev_scaled_heatmap, size * cell_size * size * cell_size * sizeof(int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpyAsync(blurred_heatmap, dev_blurred_heatmap, size * cell_size * size * cell_size * sizeof(int), hipMemcpyDeviceToHost);

	//cudaStatus = hipDeviceSynchronize();
	hipDeviceSynchronize();
	hipFree(dev_heatmap);
	hipFree(dev_scaled_heatmap);
	hipFree(dev_blurred_heatmap);
	hipFree(dev_desiredX);
	hipFree(dev_desiredY);

	/*hipHostFree(heatmap);
	hipHostFree(scaled_heatmap);
	hipHostFree(blurred_heatmap);
	hipHostFree(desiredX);
	hipHostFree(desiredY);*/

	return cudaStatus;
}

void cuda_updateHeatmap(Ped::Model *model, int *heatmap, int *scaled_heatmap, int *blurred_heatmap, int size, int cell_size, int *desiredX, int *desiredY, const int agents)
{
	hipError_t cudaStatus = createHeatmapWithCuda(model, heatmap, scaled_heatmap, blurred_heatmap, size, cell_size, desiredX, desiredY, agents);
}

int cuda_test()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

Tuple cuda_tick(const int *x, const int *y, const float *destinationX, const float *destinationY, int *desiredX, int *desiredY, const int size1)
{
	hipError_t cudaStatus = computeNextPositionWithCuda(x, y, destinationX, destinationY, size1, desiredX, desiredY);

	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "computeNextPositionWithCuda failed!");
	//	//return 1;
	//}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipDeviceReset failed!");
	//	//return 1;
	//}

	Tuple r = { desiredX, desiredY };
	return r;
}

hipError_t computeNextPositionWithCuda(const int *x, const int *y, const float *destinationX, const float *destinationY, unsigned int size, int *desiredX, int *desiredY)
{
	int *dev_x = 0;
	int *dev_y = 0;
	float *dev_destinationX = 0;
	float *dev_destinationY = 0;
	int *dev_desiredX = 0;
	int *dev_desiredY = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	fprintf(stderr, "%s.\n", hipGetErrorString(hipGetLastError()));
	goto Error;
	}*/

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(int));
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}*/

	cudaStatus = hipMalloc((void**)&dev_y, size * sizeof(int));
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}*/

	cudaStatus = hipMalloc((void**)&dev_destinationX, size * sizeof(float));
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}*/

	cudaStatus = hipMalloc((void**)&dev_destinationY, size * sizeof(float));
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}
	*/
	cudaStatus = hipMalloc((void**)&dev_desiredX, size * sizeof(int));
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}
	*/
	cudaStatus = hipMalloc((void**)&dev_desiredY, size * sizeof(int));
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}
	*/
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_x, x, size * sizeof(int), hipMemcpyHostToDevice);
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}
	*/
	cudaStatus = hipMemcpy(dev_y, y, size * sizeof(int), hipMemcpyHostToDevice);
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}*/

	cudaStatus = hipMemcpy(dev_destinationX, destinationX, size * sizeof(float), hipMemcpyHostToDevice);
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}*/

	cudaStatus = hipMemcpy(dev_destinationY, destinationY, size * sizeof(float), hipMemcpyHostToDevice);
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}
	*/
	// Launch a kernel on the GPU with one thread for each element.
	int blocks, width;
	if (size > 1024) {
		blocks = 4;
		width = size / 4;
	}
	else {
		blocks = 1;
		width = size;
	}
	computeNextPositionKernel << <blocks, width >> >(dev_desiredX, dev_desiredY, dev_x, dev_y, dev_destinationX, dev_destinationY);

	// Check for any errors launching the kernel
	//cudaStatus = hipGetLastError();
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "computeNextPositionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	goto Error;
	}
	else
	{
	fprintf(stderr, "Cuda launch succeeded! \n");
	}*/

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	// cudaStatus = hipDeviceSynchronize();
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	goto Error;
	}*/

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(desiredX, dev_desiredX, size * sizeof(int), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess) {
	//		fprintf(stderr, "hipMemcpy failed!");
	//		goto Error;
	//	}

	cudaStatus = hipMemcpy(desiredY, dev_desiredY, size * sizeof(int), hipMemcpyDeviceToHost);
	/*if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}*/

	//Error:
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_destinationX);
	hipFree(dev_destinationY);
	hipFree(dev_desiredX);
	hipFree(dev_desiredY);
	//if (cudaStatus != 0) {
	//	fprintf(stderr, "Cuda does not seem to be working properly.\n"); // This is not a good thing
	//}
	//else {
	//	//fprintf(stderr, "Cuda functionality test succeeded.\n"); // This is a good thing
	//}

	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		fprintf(stderr, "%s.\n", hipGetErrorString(hipGetLastError()));
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	else
	{
		//fprintf(stderr, "Cuda launch succeeded! \n");
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	if (cudaStatus != 0) {
		fprintf(stderr, "Cuda does not seem to be working properly.\n"); // This is not a good thing
	}
	else {
		fprintf(stderr, "Cuda functionality test succeeded.\n"); // This is a good thing
	}

	return cudaStatus;
}